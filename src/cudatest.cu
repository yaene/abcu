#include <hip/hip_runtime.h>

#include <string>

extern "C" {
__device__ __host__ void Abc_Start();
__device__ __host__ void Abc_Stop();
typedef struct Abc_Frame_t_ Abc_Frame_t;
typedef struct Abc_Ntk_t_ Abc_Ntk_t;
typedef struct Vec_Ptr_t_ Vec_Ptr_t;
typedef struct Abc_Obj_t_ Abc_Obj_t;
__device__ __host__ Abc_Frame_t* Abc_FrameGetGlobalFrame();
__device__ __host__ int Cmd_CommandExecute(Abc_Frame_t* pAbc,
                                           const char* sCommand);
__device__ __host__ Abc_Ntk_t* Abc_FrameReadNtk(Abc_Frame_t* pAbc);
}

__device__ __host__ Abc_Ntk_t* read_circuit() {
  auto abc = Abc_FrameGetGlobalFrame();
  Cmd_CommandExecute(abc, "tiny.blif");
  Cmd_CommandExecute(abc, "print_stats");
  return Abc_FrameReadNtk(abc);
}

__global__ void SimulationKernel() {
  Abc_Start();
  auto network = read_circuit();
  Abc_Stop();
}

int main(int argc, char const* argv[]) {
  SimulationKernel<<<1, 1>>>();
  return 0;
}
